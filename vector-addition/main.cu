#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// External function from solution.cu
extern "C" void solve(const float* A, const float* B, float* C, int N);

int main() {
    // Input vectors
    std::vector<float> A = {1.0f, 2.0f, 3.0f, 4.0f};
    std::vector<float> B = {5.0f, 6.0f, 7.0f, 8.0f};
    int N = A.size();

    // Output vector
    std::vector<float> C(N);

    // Device pointers
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), N * sizeof(float), hipMemcpyHostToDevice);

    // Call the vector addition function
    solve(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(C.data(), d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Vector A: [";
    for (int i = 0; i < N; i++) {
        std::cout << A[i] << (i < N-1 ? ", " : "");
    }
    std::cout << "]" << std::endl;

    std::cout << "Vector B: [";
    for (int i = 0; i < N; i++) {
        std::cout << B[i] << (i < N-1 ? ", " : "");
    }
    std::cout << "]" << std::endl;

    std::cout << "Result C: [";
    for (int i = 0; i < N; i++) {
        std::cout << C[i] << (i < N-1 ? ", " : "");
    }
    std::cout << "]" << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}